#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

#define MAX_THREADS 1024
#define BLOCK_DIM 16 // MAX: 32 since we allocate threads to blocks as BLOCK_DIM * BLOCK_DIM

/* Notes:
 - max number of blocks is 2560, or 160 if a full block of 32x32 threads is used
 - max number of threads for 1 NVIDIA V100 (has 80 SMs) is 163840
*/

typedef float dtype;

// UNUSED. reused from reduction, only for number of threads
unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

// UNUSED. reused from reduction, only to dynamically get number of threads and blocks
void getNumBlocksAndThreads(int n, int maxThreads, int &blocks, int &threads)
{
     threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
     blocks = (n + threads - 1) / threads;
}

// UNUSED! Efficient version below.
__global__ 
void matTrans_naive(dtype* AT, dtype* A, int N)  {
     /* Naive approach */
     //int end = N*N - 1;
     unsigned int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
     // unused for 1D: unsigned int yIdx = xIdx * (N * (blockIdx.x + 1)) % end;     
     //printf("N: %d, end: %d\n", N*N, end);     

     if (xIdx < N && yIdx < N) {
	unsigned int idx_in = xIdx + N * yIdx;
	unsigned int idx_out = yIdx + N * xIdx;
       // printf("in: %d, out: %d\n", idx_in, idx_out);     
	AT[idx_out] = A[idx_in];

        // unused: the very last element needs to be filled in since indices cover only up end - 2.
        //if (threadIdx.x == end)
	//   AT[end] = A[idx_in];

        //printf("AT[%d]: %f, taking in A[%d] = %f\n", idx_out, AT[idx_out], idx_in, A[idx_in]);
     }
}

// Efficient approach using shared memory and padding to avoid bank conflicts
__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
     __shared__ dtype temp[BLOCK_DIM][BLOCK_DIM+1];

     unsigned int xIdx = blockIdx.x * BLOCK_DIM + threadIdx.x;
     unsigned int yIdx = blockIdx.y * BLOCK_DIM + threadIdx.y;

     if ((xIdx < N) && (yIdx < N)) {
	unsigned int idx_in = xIdx + N * yIdx;
	temp[threadIdx.y][threadIdx.x] = A[idx_in];
     }

     __syncthreads();
     
     // transposed indices
     xIdx = blockIdx.y * BLOCK_DIM + threadIdx.x;
     yIdx = blockIdx.x * BLOCK_DIM + threadIdx.y;
     
     if ((xIdx < N) && (yIdx < N)) {
	unsigned int idx_out = xIdx + N * yIdx; // optional since idx_out = idx_in for N-by-N matrix
	AT[idx_out] = temp[threadIdx.x][threadIdx.y];
     }
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
 		//printf("GPU: %f, CPU: %f\n", a[i], b[i]); 
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  struct stopwatch_t* timer = NULL;
  long double t_gpu;
  const int mem_size = N * N * sizeof(dtype);  

  // Added: Allocate device memory
  dtype *d_A, *d_AT; // device arrays; A and AT are in host!   
  int threads, blocks;
  CUDA_CHECK_ERROR (hipMalloc (&d_A, mem_size));
  CUDA_CHECK_ERROR (hipMalloc (&d_AT, mem_size));
  CUDA_CHECK_ERROR (hipMemcpy (d_A, A, mem_size, hipMemcpyHostToDevice));
  
  // Added: Block and thread initialization	
  //getNumBlocksAndThreads(N*N, MAX_THREADS, blocks, threads);
  //dim3 gblocks(blocks, 1, 1);
  //dim3 tb(threads, 1, 1);
  blocks = N / BLOCK_DIM;
  threads = BLOCK_DIM;
  printf("2D allocation. Number of blocks: %d; number of threads: %d\n", blocks, threads); 

  dim3 gblocks(blocks, blocks, 1);
  dim3 tb(threads, threads, 1);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();
  
  // Added: warm up kernel
  matTrans <<<gblocks, tb>>> (d_AT, d_A, N);
  hipDeviceSynchronize();  

  stopwatch_start (timer);

  /* run your kernel here */
  matTrans <<<gblocks, tb>>> (d_AT, d_A, N);

  /* end kernel */   
  hipDeviceSynchronize ();
  t_gpu = stopwatch_stop (timer);
  fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );
  
  // Added: Copy result back to host, then free the device arrays
  CUDA_CHECK_ERROR (hipMemcpy (AT, d_AT, mem_size, hipMemcpyDeviceToHost)); 
  //CUDA_CHECK_ERROR (hipFree(d_A));
  //CUDA_CHECK_ERROR (hipFree(d_AT));
}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stdout, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
